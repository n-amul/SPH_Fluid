#include "hip/hip_runtime.h"
#include <glm/gtx/norm.hpp>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include <neighborTable.h>
#include <kernels/sphGPU.h>

static const float BOX_WIDTH = 10.f;
static const float BOX_COLLISION_ELASTICITY = 1.f;
static const float BOX_COLLISION_OFFSET = 0.00001;

static const uint16_t MAX_NEIGHBORS = 32;

/// Returns a hash of the cell position
__device__ uint32_t getHashDevice(const glm::ivec3 &cell)
{
    return (
        (uint)(cell.x * 73856093)
        ^ (uint)(cell.y * 19349663)
        ^ (uint)(cell.z * 83492791)
    ) % TABLE_SIZE;
}

/// Get the cell that the particle is in.
__device__ glm::ivec3 getCellDevice(Particle *p, float h)
{
    return {p->position.x / h, p->position.y / h, p->position.z / h};
}

/// Calculate particle hashes
__global__ void calculateHashesKernel(
    Particle *particles, const size_t particleCount, float h)
{
    size_t particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (particleIndex >= particleCount) {
        return;
    }
    Particle *particle = &particles[particleIndex];
    particle->hash = getHashDevice(getCellDevice(particle, h));
}

/// Comparison struct for sorting particles by hash.
struct HashComp
{
    __host__ __device__ bool operator()(
            const Particle& p1, const Particle& p2) {
        return p1.hash < p2.hash;
    }
};

/// Constructs the hash to particle map, storing the result
/// in `hashToParticleMap`.
__global__ void constructHashToParticleMap(
    Particle *sortedParticles, const size_t particleCount,
    uint32_t *hashToParticleMap)
{
    size_t particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (particleIndex >= particleCount) {
        return;
    }

    uint32_t prevHash = particleIndex == 0
        ? NO_PARTICLE : sortedParticles[particleIndex-1].hash;
    uint32_t currentHash = sortedParticles[particleIndex].hash;
    if (currentHash != prevHash) {
        hashToParticleMap[currentHash] = particleIndex;
    }
}

/// Calculates neighbors and stores the result in `neighborList`
__global__ void initNeighborList(
    Particle *particles, const size_t particleCount,
    const uint32_t *hashToParticleMap, const SPHSettings settings,
    Particle **neighborList)
{
    size_t piIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (piIndex > particleCount) {
        return;
    }
    Particle *pi = &particles[piIndex];
    glm::ivec3 cell = getCellDevice(pi, settings.h);
    size_t neighborCount = 0;
    for (int x = -1; x <= 1; x++) {
        for (int y = -1; y <= 1; y++) {
            for (int z = -1; z <= 1; z++) {
                uint16_t cellHash
                    = getHashDevice(cell + glm::ivec3(x, y, z));
                uint32_t pjIndex = hashToParticleMap[cellHash];
                if (pjIndex == NO_PARTICLE) {
                    continue;
                }
                while (pjIndex < particleCount) {
                    if (pjIndex == piIndex) {
                        pjIndex++;
                        continue;
                    }
                    Particle *pj = &particles[pjIndex];
                    if (pj->hash != cellHash) {
                        break;
                    }
                    float dist2 = glm::length2(pj->position - pi->position);
                    if (dist2 < settings.h2 && pi != pj) {
                        neighborList[piIndex * MAX_NEIGHBORS + neighborCount]
                            = pj;
                        neighborCount++;
                    }
                    pjIndex++;
                }
            }
        }
    }

    // Fill the rest of the table with null pointers
    for (size_t i = neighborCount; i < MAX_NEIGHBORS; i++) {
        neighborList[piIndex * MAX_NEIGHBORS + i] = nullptr;
    }
}

/// Kernel computation function for calculating density
/// and pressures of particles in the given SPH System.
__global__ void calculateDensitiesAndPressuresKernel(
    Particle *particles, const size_t particleCount,
    Particle **neighborList, const SPHSettings settings)
{
    size_t piIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (piIndex > particleCount) {
        return;
    }
    Particle *pi = &particles[piIndex];
    pi->density = settings.selfDens;
    glm::ivec3 cell = getCellDevice(pi, settings.h);

    size_t neighborOffset = piIndex * MAX_NEIGHBORS;
    const Particle *pj = neighborList[neighborOffset];
    while (pj) {
        float dist2 = glm::length2(pj->position - pi->position);
        pi->density += settings.massPoly6Product
                    * glm::pow(settings.h2 - dist2, 3);
        neighborOffset++;
        pj = neighborList[neighborOffset];
    }

    pi->pressure = settings.gasConstant * (pi->density - settings.restDensity);
}

/// Parallel computation function for calculating forces
/// of particles in the given SPH System.
__global__ void calculateForcesKernel(
    Particle *particles, const size_t particleCount,
    Particle **neighborList, const SPHSettings settings)
{
    size_t piIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (piIndex > particleCount) {
        return;
    }
    Particle *pi = &particles[piIndex];
    pi->force = glm::vec3(0);
    glm::ivec3 cell = getCellDevice(pi, settings.h);

    size_t neighborOffset = piIndex * MAX_NEIGHBORS;
    const Particle *pj = neighborList[neighborOffset];
    while (pj) {
        // Unit direction and length
        float dist = glm::length(pj->position - pi->position);
        glm::vec3 dir = glm::normalize(pj->position - pi->position);

        // Apply pressure force
        glm::vec3 pressureForce
            = -dir * settings.mass
              * (pi->pressure + pj->pressure) / (2 * pj->density)
              * settings.spikyGrad;
        pressureForce *= std::pow(settings.h - dist, 2);
        pi->force += pressureForce;

        // Apply viscosity force
        glm::vec3 velocityDif = pj->velocity - pi->velocity;
        glm::vec3 viscoForce
            = settings.viscosity * settings.mass
              * (velocityDif / pj->density) * settings.spikyLap
              * (settings.h - dist);
        pi->force += viscoForce;

        neighborOffset++;
        pj = neighborList[neighborOffset];
    }
}

/// Parallel computation function moving positions
/// of particles in the given SPH System.
__global__ void updateParticlePositionsKernel(
    Particle *particles, const size_t particleCount,
    glm::mat4 *particleTransforms, const SPHSettings settings,
    const float deltaTime)
{
    size_t pIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (pIndex > particleCount) {
        return;
    }
    Particle *p = &particles[pIndex];

    //calculate acceleration and velocity
    glm::vec3 acceleration
        = p->force / p->density + glm::vec3(0, settings.g, 0);
    p->velocity += acceleration * deltaTime;

    // Update position
    p->position += p->velocity * deltaTime;

    // Handle collisions with box
    if (p->position.y < settings.h) {
        p->position.y = -p->position.y + 2 * settings.h
                        + BOX_COLLISION_OFFSET;
        p->velocity.y = -p->velocity.y * BOX_COLLISION_ELASTICITY;
    }

    if (p->position.x < settings.h - BOX_WIDTH) {
        p->position.x = -p->position.x + 2 * (settings.h - BOX_WIDTH)
                        + BOX_COLLISION_OFFSET;
        p->velocity.x = -p->velocity.x * BOX_COLLISION_ELASTICITY;
    }

    if (p->position.x > -settings.h + BOX_WIDTH) {
        p->position.x = -p->position.x + 2 * -(settings.h - BOX_WIDTH)
                        - BOX_COLLISION_OFFSET;
        p->velocity.x = -p->velocity.x * BOX_COLLISION_ELASTICITY;
    }

    if (p->position.z < settings.h - BOX_WIDTH) {
        p->position.z = -p->position.z + 2 * (settings.h - BOX_WIDTH)
                        + BOX_COLLISION_OFFSET;
        p->velocity.z = -p->velocity.z * BOX_COLLISION_ELASTICITY;
    }

    if (p->position.z > -settings.h + BOX_WIDTH) {
        p->position.z = -p->position.z + 2 * -(settings.h - BOX_WIDTH)
                        - BOX_COLLISION_OFFSET;
        p->velocity.z = -p->velocity.z * BOX_COLLISION_ELASTICITY;
    }

    particleTransforms[pIndex]
        = glm::translate(p->position) * settings.sphereScale;
}

void updateParticlesGPU(
    Particle *particles, glm::mat4 *particleTransforms,
    const size_t particleCount, const SPHSettings &settings,
    float deltaTime)
{
    size_t threadsPerBlock = 512;
    size_t gridSize = particleCount / threadsPerBlock + 1;

    // Start by copying particles to GPU.
    thrust::device_vector<Particle> dParticleVector(
        particles, particles + particleCount);
    Particle *dParticles = thrust::raw_pointer_cast(dParticleVector.data());

    // Set up hashes for all particles
    calculateHashesKernel<<<gridSize, threadsPerBlock>>>(
        dParticles, particleCount, settings.h);
    hipDeviceSynchronize();

    // Sort particles by hash.
    thrust::sort(
        dParticleVector.begin(), dParticleVector.end(), HashComp());

    // Set up the hash->particle map.
    thrust::device_vector<uint32_t> dHashToParticleMapVector(
        TABLE_SIZE, NO_PARTICLE);
    uint32_t *dHashToParticleMap
        = thrust::raw_pointer_cast(dHashToParticleMapVector.data());
    constructHashToParticleMap<<<gridSize, threadsPerBlock>>>(
        dParticles, particleCount, dHashToParticleMap);
    hipDeviceSynchronize();

    // Calculate neighbors for every particle.
    Particle **dNeighborList;
    hipMalloc((void**)&dNeighborList,
               sizeof(Particle *) * particleCount * MAX_NEIGHBORS);
    initNeighborList<<<gridSize, threadsPerBlock>>>(
        dParticles, particleCount, dHashToParticleMap, settings,
        dNeighborList);
    hipDeviceSynchronize();

    // Calculate densities and pressures.
    calculateDensitiesAndPressuresKernel<<<gridSize, threadsPerBlock>>>(
        dParticles, particleCount, dNeighborList, settings);
    hipDeviceSynchronize();

    // Calculate forces.
    calculateForcesKernel<<<gridSize, threadsPerBlock>>>(
        dParticles, particleCount, dNeighborList, settings);
    hipDeviceSynchronize();

    hipFree(dNeighborList);

    // Update positions and transforms.
    glm::mat4 *dParticleTransforms;
    size_t transformsSize = sizeof(glm::mat4) * particleCount;
    hipMalloc((void**)&dParticleTransforms, transformsSize);
    updateParticlePositionsKernel<<<gridSize, threadsPerBlock>>>(
        dParticles, particleCount, dParticleTransforms, settings, deltaTime);
    hipDeviceSynchronize();

    hipMemcpy(particles, dParticles, sizeof(Particle) * particleCount,
               hipMemcpyDeviceToHost);
    hipMemcpy(particleTransforms, dParticleTransforms, transformsSize,
               hipMemcpyDeviceToHost);
    hipFree(dParticleTransforms);
}